
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <pthread.h>

const int N = 1 << 27;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

void *thread(void *args)
{
    int * thread_data = (int*) args;
    const int num_streams = 8;

    hipStream_t streams[num_streams];
    float *data[num_streams];

    int deviceNum; 
    hipError_t ret = hipGetDeviceCount(&deviceNum);
    ret = hipSetDevice(*thread_data);
    printf("device num: %d\n", *thread_data);

    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipError_t  ret0 = hipMalloc(&data[i], N * sizeof(float));

        if (ret0 != hipSuccess) {
           printf("allocate failed\n");
           return 0;
        } else {
           printf("%d MB\n", N*sizeof(float)/1024/1024);
        }

        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);

        printf("finished stream syn %d\n", i);
    }

    printf("finished all stream %d\n");
    hipDeviceReset();
    printf("finished device reset %d\n");

    return 0;
}

int main() {
  pthread_t threads[4];

  int thread_data[4];
  for(int t=0;t<4;t++){

    printf("In main: creating thread %ld\n", t);
    thread_data[t] = t;
    int rc = pthread_create(&threads[t], NULL, thread, &thread_data[t]);
    if (rc){

      printf("ERROR; return code from pthread_create() is %d\n", rc);
      exit(-1);
    }
  }

  for(int i = 0; i < 4; i++) 
    pthread_join(threads[i], NULL);

  /* Last thing that main() should do */
  pthread_exit(NULL);
}
